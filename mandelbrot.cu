#include "hip/hip_runtime.h"
#include "mandelbrot.h"
#include "config.h"
#include "logger.h"

extern "C" {
#include <stdlib.h>
#include <string.h>
}

typedef struct {
	int w;
	int h;
	int *rgb_data;
} MandelBuffer;

MandelBuffer mandelbuffer;

__device__
int getIterations(float x0, float y0, float escape_rad) {
	int iteration = 0;
	float x = 0.0;
	float y = 0.0;

	while(x*x + y*y <= escape_rad * escape_rad && iteration < MAX_ITERATIONS) {
		float tmpx = x * x - y * y + x0;
		y = 2 * x * y + y0;
		x = tmpx;
		iteration++;
	}
	return iteration;
}

__device__
int iterationsToColor(int iterations) {
	if(iterations >= MAX_ITERATIONS)
		return 0x000000; // Black

	float hue = (int)(sqrt((float)iterations) * 12.0);
	float C = 1.0;
	float X = hue / 60.0;
	X = X - (int)X;
	float r, g, b;
	if(hue >= 0.0 && hue < 60.0) {
		r = C; g = X; b = 0;
	} else if(hue >= 60.0 && hue < 120.0) {
		r = 1 - X; g = C; b = 0;
	} else if(hue >= 120.0 && hue < 180.0) {
		r = 0; g = C; b = X;
	} else if(hue >= 180.0 && hue < 240.0) {
		r = 0; g = 1 - X; b = C;
	} else if(hue >= 240.0 && hue < 300.0) {
		r = X; g = 0; b = C;
	} else {
		r = C; g = 0; b = 1 - X;
	}
	r *= 255;
	g *= 255;
	b *= 255;
	return (int)r + (int)g * 256 + (int)b * 65536;
}

__global__
void mandelbrot(int pix_w, int pix_h, float coord_x, float coord_y,
				float coord_w, float coord_h, float escape_rad, int *out) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < pix_w * pix_h; i += stride) {
		float cx = (float)(i % pix_w);
		float cy = (float)(i / pix_w);
		cx = cx / (float)pix_w * coord_w + coord_x;
		cy = cy / (float)pix_h * coord_h + coord_y;

		int iters = getIterations(cx, cy, escape_rad);
		int color = iterationsToColor(iters);
		out[i] = 0xff000000 | color; // Write color with full alpha into output
    }
}

__host__
inline int round_simple(float f) {
	return (int)(f + 0.5);
}

__host__
int blend(int in_color, int blend_color, float ratio) {
	float r_in = in_color & 0xff;
	float g_in = (in_color & 0xff00) >> 8;
	float b_in = (in_color & 0xff0000) >> 16;

	float r_blend = blend_color & 0xff;
	float g_blend = (blend_color & 0xff00) >> 8;
	float b_blend = (blend_color & 0xff0000) >> 16;

	r_in = r_in * (1.0 - ratio) + r_blend * ratio;
	g_in = g_in * (1.0 - ratio) + g_blend * ratio;
	b_in = b_in * (1.0 - ratio) + b_blend * ratio;

	return (int)r_in + (int)g_in * 256 + (int)b_in * 65536;
}

__host__
int clamp(int i, int min, int max) {
	return i < min ? min : i > max ? max : i;
}

__host__
void scaleLinear(int w_in, int h_in, int w_out, int h_out, int *in_rgb,
		int *out_rgb) {
	float scale_x = (float)w_in / (float)w_out;
	float scale_y = (float)h_in / (float)h_out;

	for(int out_y = 0; out_y < h_out; out_y++) {
		float in_y = scale_y * (float)out_y;
		for(int out_x = 0; out_x < w_out; out_x++) {
			float in_x = scale_x * (float)out_x;

			float px = in_x - (int)in_x;
			float py = in_y - (int)in_y;

			int x = (int)in_x;
			int y = (int)in_y;

			float p00, p01, p10, p11;
			p00 = px * py;
			p01 = (1.0 - px) * py;
			p10 = px * (1.0 - py);
			p11 = (1.0 - px) * (1.0 - py);

			int color = 0;
			color = blend(color, in_rgb[clamp(y, 0, h_in) * w_in +
					clamp(x, 0, w_in)], p00);
			color = blend(color, in_rgb[clamp(y, 0, h_in) * w_in +
					clamp(x + 1, 0, w_in)], p01);
			color = blend(color, in_rgb[clamp(y + 1, 0, h_in) * w_in +
					clamp(x, 0, w_in)], p10);
			color = blend(color, in_rgb[clamp(y + 1, 0, h_in) * w_in +
					clamp(x + 1, 0, w_in)], p11);

			out_rgb[out_y * w_out + out_x] = 0xff000000 | color;
		}
	}
}

__host__
void scaleNN(int w_in, int h_in, int w_out, int h_out, int *in_rgb,
		int *out_rgb) {
	float scale_x = (float)w_in / (float)w_out;
	float scale_y = (float)h_in / (float)h_out;

	for(int y = 0; y < h_out; y++) {
		int in_y = round_simple(scale_y * (float)y);
		for(int x = 0; x < w_out; x++) {
			int in_x = round_simple(scale_x * (float)x);
			out_rgb[y * w_out + x] = in_rgb[in_y * w_in + in_x];
		}
	}
}

int get_device_attributes() {

	int ret;
	int cur_device;

	ret = hipGetDevice(&cur_device);
	if(ret != hipSuccess) goto error;
	log(DEBUG, "Running on device %d\n", cur_device);

	hipDeviceProp_t props;
	ret = hipGetDeviceProperties(&props, cur_device);
	if(ret != hipSuccess) goto error;
	log(DEBUG, "Running on GPU %s\n", props.name);
	log(DEBUG, "Device Cuda Version: %d.%d\n", props.major, props.minor);
	log(DEBUG, "Managed memory is%s supported\n",
			props.managedMemory ? "" : " not");

	return 0;
error:
	return -1;
}

int mandelbrotInit(int w, int h) {
	log(INFO, "Starting Mandelbrot Engine...\n");
	int *img_data = NULL;
	int ret = hipMallocManaged(&img_data, w * h * sizeof(int));
	if(ret != hipSuccess) goto error;
	mandelbuffer = {w, h, img_data};

	ret = get_device_attributes();
	if(ret != hipSuccess) goto error;

	return 0;
error:
	if(img_data != NULL)
		hipFree(img_data);
	return ret;
}

void mandelbrotCleanup() {
	hipFree(mandelbuffer.rgb_data);
}

void generateImage(Rectangle coord_rect, int *out_argb) {
	if(out_argb == NULL)
		return;

	mandelbrot<<<RENDER_THREAD_BLOCKS, RENDER_THREADS>>>(mandelbuffer.w, mandelbuffer.h,
			coord_rect.x, coord_rect.y,
			coord_rect.w, coord_rect.h, ESCAPE_RADIUS, mandelbuffer.rgb_data);
	hipDeviceSynchronize();

	memcpy(out_argb, mandelbuffer.rgb_data,
			mandelbuffer.w * mandelbuffer.h * sizeof(int));
}

void generateImage2(int w, int h, Rectangle coord_rect, int *out_argb) {
	if(out_argb == NULL)
		return;

	int *out;
	hipMallocManaged(&out, w * h * sizeof(int));
	mandelbrot<<<RENDER_THREAD_BLOCKS, RENDER_THREADS>>>(w, h, coord_rect.x, coord_rect.y,
			coord_rect.w, coord_rect.h, ESCAPE_RADIUS, out);
	hipDeviceSynchronize();

	memcpy(out_argb, out, w * h * sizeof(int));
	hipFree(out);
}

// aa_counter defines the shift and blend percentage
void doAntiAlias(Rectangle coord_rect, int *argb_buf, int aa_counter) {
	if(argb_buf == NULL)
		return;

	float shift_amount_x = coord_rect.w / (float)mandelbuffer.w / 3.0;
	float shift_amount_y = coord_rect.h / (float)mandelbuffer.h / 3.0;

	float shift_x = coord_rect.x +
			((aa_counter & 2) ? 1.0 : -1.0) * shift_amount_x;
	float shift_y = coord_rect.y +
			((aa_counter & 1) ? 1.0 : -1.0) * shift_amount_y;

	mandelbrot<<<RENDER_THREAD_BLOCKS, RENDER_THREADS>>>(mandelbuffer.w, mandelbuffer.h,
			shift_x, shift_y,
			coord_rect.w, coord_rect.h, ESCAPE_RADIUS, mandelbuffer.rgb_data);
	hipDeviceSynchronize();

	aa_counter += 2;

	// blend them together
	for(int i = 0; i < mandelbuffer.w * mandelbuffer.h; i++) {
		int blend_color = blend(argb_buf[i],
				mandelbuffer.rgb_data[i], 1.0 / aa_counter);
		argb_buf[i] = 0xff000000 | blend_color; // apply full alpha
	}
}

void scaleImage(int w_in, int h_in, int w_out, int h_out, int *in_rgb,
		int *out_rgb, int interp_method) {

	switch(interp_method) {
		case INTERP_LINEAR:
			scaleLinear(w_in, h_in, w_out, h_out, in_rgb, out_rgb);
			break;
		case INTERP_NN:
			/* FALLTHRU */
		default:
			scaleNN(w_in, h_in, w_out, h_out, in_rgb, out_rgb);
	}
}

