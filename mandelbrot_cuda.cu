#include "hip/hip_runtime.h"
#include "mandelbrot_cuda.h"
#include "config.h"
#include "logger.h"

#include "util.h"

extern "C" {
#include <stdlib.h>
#include <string.h>
}

MandelBuffer mandelbuffer;

int max_iterations = DEFAULT_ITERATIONS;

__device__
int getIterations(float x0, float y0, float escape_rad, int max_iters) {
	int iteration = 0;
	float x = 0.0;
	float y = 0.0;

	while(x*x + y*y <= escape_rad * escape_rad && iteration < max_iters) {
		float tmpx = x * x - y * y + x0;
		y = 2 * x * y + y0;
		x = tmpx;
		iteration++;
	}
	return iteration;
}

__device__
int iterationsToColor(int iterations, int max_iters) {
	if(iterations >= max_iters)
		return 0x000000; // Black

	float hue = (int)(log2((float)iterations) * 12.0);
	float C = 1.0;
	float X = hue / 60.0;
	X = X - (int)X;
	float r, g, b;
	if(hue >= 0.0 && hue < 60.0) {
		r = C; g = X; b = 0;
	} else if(hue >= 60.0 && hue < 120.0) {
		r = 1 - X; g = C; b = 0;
	} else if(hue >= 120.0 && hue < 180.0) {
		r = 0; g = C; b = X;
	} else if(hue >= 180.0 && hue < 240.0) {
		r = 0; g = 1 - X; b = C;
	} else if(hue >= 240.0 && hue < 300.0) {
		r = X; g = 0; b = C;
	} else {
		r = C; g = 0; b = 1 - X;
	}
	r *= 255;
	g *= 255;
	b *= 255;
	return (int)r + (int)g * 256 + (int)b * 65536;
}

__global__
void mandelbrot(int pix_w, int pix_h, float coord_x, float coord_y,
				float coord_w, float coord_h, float escape_rad,
				int *out, int max_iters) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < pix_w * pix_h; i += stride) {
		float cx = (float)(i % pix_w);
		float cy = (float)(i / pix_w);
		cx = cx / (float)pix_w * coord_w + coord_x;
		cy = cy / (float)pix_h * coord_h + coord_y;

		int iters = getIterations(cx, cy, escape_rad, max_iters);
		int color = iterationsToColor(iters, max_iters);
		out[i] = 0xff000000 | color; // Write color with full alpha into output
    }
}

__host__
void changeIterationsCuda(int diff) {
	int new_iters = clamp(max_iterations + diff, 1, 5000);
	log(INFO, "Changing Maximum Iterations to %d\n", new_iters);
	max_iterations = new_iters;
}

int getDeviceAttributes() {
	int ret;
	int cur_device;

	ret = hipGetDevice(&cur_device);
	if(ret != hipSuccess) goto error;
	log(DEBUG, "Running on device %d\n", cur_device);

	hipDeviceProp_t props;
	ret = hipGetDeviceProperties(&props, cur_device);
	if(ret != hipSuccess) goto error;
	log(DEBUG, "Device name: %s\n", props.name);
	log(DEBUG, "Device Cuda Version: %d.%d\n", props.major, props.minor);
	log(DEBUG, "Managed memory is%s supported\n",
			props.managedMemory ? "" : " not");

	return 0;
error:
	return -1;
}

int mandelbrotCudaInit(int w, int h) {
	log(VERBOSE, "Starting Mandelbrot Engine...\n");
	int *img_data = NULL;
	int ret = hipMallocManaged(&img_data, w * h * sizeof(int));
	if(ret != hipSuccess) goto error;
	mandelbuffer = {w, h, img_data};

	ret = getDeviceAttributes();
	if(ret != hipSuccess) goto error;

	return 0;
error:
	if(img_data != NULL)
		hipFree(img_data);
	return ret;
}

void mandelbrotCudaCleanup() {
	log(VERBOSE, "Cleaning up Mandelbrot Engine...\n");
	hipFree(mandelbuffer.rgb_data);
}

void generateImageCuda(Rectangle coord_rect, int *out_argb) {
	if(out_argb == NULL)
		return;

	mandelbrot<<<RENDER_THREAD_BLOCKS, RENDER_THREADS>>>(mandelbuffer.w, mandelbuffer.h,
			coord_rect.x, coord_rect.y,
			coord_rect.w, coord_rect.h, ESCAPE_RADIUS, mandelbuffer.rgb_data, max_iterations);
	hipDeviceSynchronize();

	memcpy(out_argb, mandelbuffer.rgb_data,
			mandelbuffer.w * mandelbuffer.h * sizeof(int));
}

void generateImageCudaWH(int w, int h, Rectangle coord_rect, int *out_argb) {
	if(out_argb == NULL)
		return;

	int *out;
	hipMallocManaged(&out, w * h * sizeof(int));
	mandelbrot<<<RENDER_THREAD_BLOCKS, RENDER_THREADS>>>(w, h, coord_rect.x, coord_rect.y,
			coord_rect.w, coord_rect.h, ESCAPE_RADIUS, out, max_iterations);
	hipDeviceSynchronize();

	memcpy(out_argb, out, w * h * sizeof(int));
	hipFree(out);
}

// aa_counter defines the shift and blend percentage
void doAntiAliasCuda(Rectangle coord_rect, int *argb_buf, int aa_counter) {
	if(argb_buf == NULL)
		return;
	if(aa_counter < 0 || aa_counter > 3)
		return;

	float shift_amount_x = coord_rect.w / (float)mandelbuffer.w / 3.0;
	float shift_amount_y = coord_rect.h / (float)mandelbuffer.h / 3.0;

	// Use the last two bits of aa_counter as shift indicator
	// For now this works fine, as the aa_counter is only allowed in a range of 0 to 3
	float shift_x = coord_rect.x +
			((aa_counter & 2) ? 1.0 : -1.0) * shift_amount_x;
	float shift_y = coord_rect.y +
			((aa_counter & 1) ? 1.0 : -1.0) * shift_amount_y;

	mandelbrot<<<RENDER_THREAD_BLOCKS, RENDER_THREADS>>>(mandelbuffer.w, mandelbuffer.h,
			shift_x, shift_y,
			coord_rect.w, coord_rect.h, ESCAPE_RADIUS, mandelbuffer.rgb_data, max_iterations);
	hipDeviceSynchronize();

	aa_counter += 2;

	// blend them together
	for(int i = 0; i < mandelbuffer.w * mandelbuffer.h; i++) {
		int blend_color = blend(argb_buf[i],
				mandelbuffer.rgb_data[i], 1.0 / aa_counter);
		argb_buf[i] = 0xff000000 | blend_color; // apply full alpha
	}
}

