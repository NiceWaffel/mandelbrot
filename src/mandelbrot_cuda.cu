#include "hip/hip_runtime.h"
#include "mandelbrot_cuda.h"

extern "C" {
#include "util.h"
#include "logger.h"
#include "config.h"

#include <stdlib.h>
#include <string.h>
}

MandelBuffer mandelbuffer;

int max_iterations = DEFAULT_ITERATIONS;

__device__
int getIterations(float x0, float y0, float escape_rad, int max_iters) {
	int iteration = 0;
	float x = 0.0;
	float y = 0.0;

	while(x*x + y*y <= escape_rad * escape_rad && iteration < max_iters) {
		float tmpx = x * x - y * y + x0;
		y = 2 * x * y + y0;
		x = tmpx;
		iteration++;
	}
	return iteration;
}

__device__
int iterationsToColor(int iterations, int max_iters) {
	if(iterations >= max_iters)
		return 0x000000; // Black

	float hue = (int)(log2((float)iterations) * 20.0);
	float C = 1.0;
	float X = hue / 60.0;
	X = X - (int)X;
	float r, g, b;
	if(hue >= 0.0 && hue < 60.0) {
		r = C; g = X; b = 0;
	} else if(hue >= 60.0 && hue < 120.0) {
		r = 1 - X; g = C; b = 0;
	} else if(hue >= 120.0 && hue < 180.0) {
		r = 0; g = C; b = X;
	} else if(hue >= 180.0 && hue < 240.0) {
		r = 0; g = 1 - X; b = C;
	} else if(hue >= 240.0 && hue < 300.0) {
		r = X; g = 0; b = C;
	} else {
		r = C; g = 0; b = 1 - X;
	}
	r *= 255;
	g *= 255;
	b *= 255;
	return (int)r + (int)g * 256 + (int)b * 65536;
}

__global__
void mandelbrot(int pix_w, int pix_h, float coord_x, float coord_y,
				float coord_w, float coord_h, float escape_rad,
				int *out, int max_iters) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < pix_w * pix_h; i += stride) {
		float cx = (float)(i % pix_w);
		float cy = (float)(i / pix_w);
		cx = cx / (float)pix_w * coord_w + coord_x;
		cy = cy / (float)pix_h * coord_h + coord_y;

		int iters = getIterations(cx, cy, escape_rad, max_iters);
		int color = iterationsToColor(iters, max_iters);
		out[i] = 0xff000000 | color; // Write color with full alpha into output
    }
}

__host__
void changeIterationsCuda(int diff) {
	int new_iters = clamp(max_iterations + diff, 1, 5000);
	mandelLog(INFO, "Changing Maximum Iterations to %d\n", new_iters);
	max_iterations = new_iters;
}

int getDeviceAttributes() {
	int ret;
	int cur_device;

	ret = hipGetDevice(&cur_device);
	if(ret != hipSuccess) goto error;
	mandelLog(DEBUG, "Running on device %d\n", cur_device);

	hipDeviceProp_t props;
	ret = hipGetDeviceProperties(&props, cur_device);
	if(ret != hipSuccess) goto error;
	mandelLog(DEBUG, "Device name: %s\n", props.name);
	mandelLog(DEBUG, "Device Cuda Version: %d.%d\n", props.major, props.minor);
	mandelLog(DEBUG, "Managed memory is%s supported\n",
			props.managedMemory ? "" : " not");

	return 0;
error:
	return -1;
}

int mandelbrotCudaInit(int w, int h) {
	mandelLog(VERBOSE, "Starting Mandelbrot Engine...\n");
	int *img_data = NULL;
	int ret = hipMallocManaged(&img_data, w * h * sizeof(int));
	if(ret != hipSuccess) goto error;
	mandelbuffer = {w, h, img_data};

	ret = getDeviceAttributes();
	if(ret != hipSuccess) goto error;

	return 0;
error:
	if(img_data != NULL)
		hipFree(img_data);
	return ret;
}

void mandelbrotCudaCleanup() {
	mandelLog(VERBOSE, "Cleaning up Mandelbrot Engine...\n");
	hipFree(mandelbuffer.rgb_data);
}

int resizeFramebufferCuda(int new_w, int new_h) {
	hipFree(mandelbuffer.rgb_data);
	int *img_data = NULL;
	if(hipMallocManaged(&img_data, new_w * new_h * sizeof(int)) != hipSuccess) {
		return -1;
	}
	mandelbuffer = {new_w, new_h, img_data};
	return 0;
}

void generateImageCuda(Rectangle coord_rect, int *out_argb) {
	if(out_argb == NULL)
		return;

	mandelbrot<<<RENDER_THREAD_BLOCKS, RENDER_THREADS>>>(mandelbuffer.w, mandelbuffer.h,
			coord_rect.x, coord_rect.y,
			coord_rect.w, coord_rect.h, ESCAPE_RADIUS, mandelbuffer.rgb_data, max_iterations);
	hipDeviceSynchronize();

	memcpy(out_argb, mandelbuffer.rgb_data,
			mandelbuffer.w * mandelbuffer.h * sizeof(int));
}

void generateImageCudaWH(int w, int h, Rectangle coord_rect, int *out_argb) {
	if(out_argb == NULL)
		return;

	int *out;
	hipMallocManaged(&out, w * h * sizeof(int));
	mandelbrot<<<RENDER_THREAD_BLOCKS, RENDER_THREADS>>>(w, h, coord_rect.x, coord_rect.y,
			coord_rect.w, coord_rect.h, ESCAPE_RADIUS, out, max_iterations);
	hipDeviceSynchronize();

	memcpy(out_argb, out, w * h * sizeof(int));
	hipFree(out);
}

// aa_counter defines the shift and blend percentage
void doAntiAliasCuda(Rectangle coord_rect, int *argb_buf, int aa_counter) {
	if(argb_buf == NULL)
		return;
	if(aa_counter < 0 || aa_counter > 7)
		return;

	float shift_amount_x, shift_amount_y;
	float shift_x, shift_y;
	if(aa_counter < 4) {
		shift_amount_x = coord_rect.w / (float)mandelbuffer.w / 3.0;
		shift_amount_y = coord_rect.h / (float)mandelbuffer.h / 3.0;

		/* Go for every corner by using bit pattern of last two bits */
		shift_x = coord_rect.x + ((aa_counter & 2) ? 1.0 : -1.0) * shift_amount_x;
		shift_y = coord_rect.y + ((aa_counter & 1) ? 1.0 : -1.0) * shift_amount_y;
	}
	else if(aa_counter < 8) {
		shift_amount_x = coord_rect.w / (float)mandelbuffer.w / 2.0;
		shift_amount_y = coord_rect.h / (float)mandelbuffer.h / 2.0;

		/*
		 * When aa_counter is:
		 * - 4: We shift in positive x direction
		 * - 5: We shift in positive y direction
		 * - 6: We shift in negative x direction
		 * - 7: We shift in negative y direction
		 */
		int even = aa_counter % 2 == 0;
		shift_x = coord_rect.x + ( even ? 1.0 : 0.0) *
				shift_amount_x * (aa_counter > 5 ? -1.0 : 1.0);
		shift_y = coord_rect.y + (!even ? 1.0 : 0.0) *
				shift_amount_y * (aa_counter > 5 ? -1.0 : 1.0);
	}

	mandelbrot<<<RENDER_THREAD_BLOCKS, RENDER_THREADS>>>(mandelbuffer.w, mandelbuffer.h,
			shift_x, shift_y,
			coord_rect.w, coord_rect.h, ESCAPE_RADIUS, mandelbuffer.rgb_data, max_iterations);
	hipDeviceSynchronize();

	aa_counter += 2;

	// blend them together
	for(int i = 0; i < mandelbuffer.w * mandelbuffer.h; i++) {
		int blend_color = blend(argb_buf[i],
				mandelbuffer.rgb_data[i], 1.0 / aa_counter);
		argb_buf[i] = 0xff000000 | blend_color; // apply full alpha
	}
}
